#include "hip/hip_runtime.h"
#include "gpuMain.h"
#include "kernels.h"

#define LEVELS 5
#define THRESH 2

float * fillArray(int n)
{
   int i;

   float *ret = (float *) malloc(sizeof(float) * n );

    for( i = 0; i < n ; i++ ) {
      ret[i] = (float) 3.3;
   }

   return ret;
}

void printArray(float *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%f ", arr[i]);

   printf("\n");
}

float cpuReduce(float * h_in, int n)
{
   double total = 0.0;

	 int i;

    for(i = 0; i < n; i ++)
        total += (double) h_in[i];
   // printf("CPU---> %fn", total);
    return total;
}

void reverseIt(float * histogram)
{
	const int n = 60;
  	float d[n];

  	float *d_hist;
  	hipMalloc(&d_hist, n * sizeof(float)); 

	  hipMemcpy(d_hist, histogram, n*sizeof(float), hipMemcpyHostToDevice);


	  staticReverse<<< 1,64 >>>(d_hist, n);
    
	  hipMemcpy(d, d_hist, n*sizeof(float), hipMemcpyDeviceToHost);

    printf("*************************************************\n");

	 for (int i = 0; i < n; i++) {
      printf("%d) %f\n",i, d[i]);
    }

    hipFree(d_hist);
}



/*
  This function and it's kernel are probably obsolete at this point, but leaving them in just in case.

  Obsolete because it requires a hueFrame of only the search window hues and not the entire frame.
  Limited, in that the frame must be reset between iterations, I'd have to remove the cudaMallocs and reuse a statically sized
  search window to make use of this. It also would perclude the opportunity of extending for multiple objects
*/

int launchMeanShiftKernelForSubFrame(unsigned char * hueFrame, int hueFrameLength, int width, int xOffset, int yOffset, int * cx, int * cy)
{
  printf("\nInside Launching GPU MeanShift...\n");

 
   unsigned char * d_in;

    hipError_t err = hipMalloc((void **)&d_in, hueFrameLength * sizeof(unsigned char)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    err = hipMemcpy(d_in, hueFrame, hueFrameLength * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipEvent_t launch_begin, launch_end;
    int tile_width = 1024;
    int num_block = ceil( (float) hueFrameLength / (float) tile_width);
    dim3 block(tile_width, 1, 1);
    dim3 grid(num_block, 1, 1);

    //Make d_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * d_out;
    err = hipMalloc((void **)&d_out, 3 * num_block * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

      int * readyArray;
    err = hipMalloc((void **)&readyArray, num_block * sizeof(int)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));
      hipMemset(readyArray, 0, sizeof(int) * num_block ); 

    //Make h_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * h_out = (float *) malloc(3 * num_block * sizeof(float));

  printf("Num_block: %d vs tile_width %d\n", num_block, tile_width);


    if(num_block <= tile_width){

     hipEventCreate(&launch_begin);
     hipEventCreate(&launch_end);

     hipEventRecord(launch_begin,0);

    gpuMeanShiftKernelForSubFrame<<< grid, block >>>(d_in, d_out, readyArray, hueFrameLength, num_block, width, xOffset, yOffset);
      

    err =  hipMemcpy(h_out, d_out, 3 * num_block * sizeof(float), hipMemcpyDeviceToHost);


     hipEventRecord(launch_end,0);
    hipEventSynchronize(launch_end);

    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    printf("GPU time cost in milliseconds for improved meanshift kernel with atomic add: %f\n", time);
    printf("improved meanshift kernel with atomic add total: M00 = %f M1x = %f M1y = %f \n", h_out[0], h_out[num_block], h_out[num_block * 2]);

    //cpuReduce(h_out, num_block);

   //  printArray(h_out, num_block);

  }
  else
    printf("Cannot launch kernel: num_block (%d) > tile_width (%d)\n", num_block, tile_width);


    hipFree(d_out);
    hipFree(readyArray);
    free(h_out);
    hipFree(d_in);

    return 1;
}


//***********************************************************************************************//
// Below launches new improved kernel stuff

//wrapper function because constant memory must be in the same file that accesses it, linking issue
void mainConstantMemoryHistogramLoad(float * histogram)
{
  hipDeviceReset();
  setConstantMemoryHistogram(histogram);
}

void initDeviceStruct(d_struct * ds, unsigned char * frame, int frameLength, int * cx, int * cy, int * col_offset, int * row_offset)
{
    hipError_t err; 
    int * d_cx;
    int * d_cy;
    int * d_col_offset;
    int * d_row_offset;
    unsigned char * d_frame;

    if(( err = hipMalloc((void **)&d_frame, frameLength * sizeof(unsigned char))) != hipSuccess)
          printf("%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_frame, frame, frameLength * sizeof(unsigned char), hipMemcpyHostToDevice);  
    if((err = hipMalloc((void **)&d_cx, sizeof(int))) != hipSuccess) 
          printf("%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_cx, cx, sizeof(int), hipMemcpyHostToDevice);
    if((err = hipMalloc((void **)&d_cy, sizeof(int))) != hipSuccess) 
          printf("%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_cy, cy, sizeof(int), hipMemcpyHostToDevice);
    if((err = hipMalloc((void **)&d_row_offset, sizeof(int))) != hipSuccess) 
          printf("%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_row_offset, row_offset, sizeof(int), hipMemcpyHostToDevice);
    if((err = hipMalloc((void **)&d_col_offset, sizeof(int))) != hipSuccess) 
          printf("%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_col_offset, col_offset, sizeof(int), hipMemcpyHostToDevice);

    (*ds).d_frame = d_frame;
    (*ds).d_cx = d_cx;
    (*ds).d_cy = d_cy;
    (*ds).d_col_offset = d_col_offset;
    (*ds).d_row_offset = d_row_offset;
}

void freeDeviceStruct(d_struct * ds)
{
    hipFree((*ds).d_frame);
    hipFree((*ds).d_cx);
    hipFree((*ds).d_cy);
    hipFree((*ds).d_row_offset);
    hipFree((*ds).d_col_offset);
}


float launchTwoKernelReduction(d_struct ds, unsigned char * frame, int frameLength, int subFrameLength, int abs_width, int sub_width, int sub_height, int * cx, int * cy, bool shouldPrint)
{
    float time = 0;
    // printf("\nInside Launching GPU MeanShift for entire frame...\n");
    hipEvent_t launch_begin, launch_end;
    int tile_width = 1024;
    int num_block = ceil( (float) subFrameLength / (float) tile_width);
    dim3 block(tile_width, 1, 1);
    dim3 grid(num_block, 1, 1);
    unsigned int dynamic_sharedMem_size = 3 * num_block * sizeof(float);

    hipError_t err; 

    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);
    hipEventRecord(launch_begin,0);
    int * h_cx = (int *) malloc(sizeof(int));
    int * h_cy = (int *) malloc(sizeof(int));
    h_cx[0] = -1;
    h_cy[0] = -1;
    //Make d_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * d_out;
    if((err = hipMalloc((void **)&d_out, 3 * num_block * sizeof(float)))!= hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    //Make h_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * h_out = (float *) malloc(3 * num_block * sizeof(float));

     int prevX = 0;
     int prevY = 0;

    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);
    hipEventRecord(launch_begin,0);


    err = hipMemcpy(ds.d_frame, frame, frameLength * sizeof(unsigned char), hipMemcpyHostToDevice);

    if(num_block <= tile_width)
    {

    while(prevX != h_cx[0] && prevY != h_cy[1]){

      prevX = h_cx[0];
      prevY = h_cy[0];

      gpuBlockReduce<<< grid, block >>>(ds.d_frame, d_out, subFrameLength, num_block, abs_width, sub_width, sub_height, ds.d_row_offset, ds.d_col_offset);
      gpuFinalReduce<<< 1, num_block, dynamic_sharedMem_size >>>(d_out, ds.d_cx, ds.d_cy, ds.d_row_offset, ds.d_col_offset, sub_width, sub_height, num_block);

      err =  hipMemcpy(h_cx, ds.d_cx, sizeof(int), hipMemcpyDeviceToHost);
      err =  hipMemcpy(h_cy, ds.d_cy, sizeof(int), hipMemcpyDeviceToHost);
     
      if(shouldPrint)
     	printf("PrevX vs NewX(%d, %d) and PrevY vs NewY(%d, %d)\n", prevX, h_cx[0], prevY, h_cy[0]);

    }
    hipDeviceSynchronize();
    hipEventRecord(launch_end,0);
    hipEventSynchronize(launch_end);
    hipEventElapsedTime(&time, launch_begin, launch_end);
  }
  else
    printf("Cannot launch kernel: num_block (%d) > tile_width (%d)\n", num_block, tile_width);

    *cx = h_cx[0];
    *cy = h_cy[0];

    hipFree(d_out);
    free(h_out);
    free(h_cx);
    free(h_cy);

    return time;
}
