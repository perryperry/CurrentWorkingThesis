#include "hip/hip_runtime.h"

#include "timing.c"
#include "gpuMain.h"
#include "gpuMerge.h"

#define LEVELS 5
#define THRESH 2

float * fillArray(int n)
{
   int i;

   float *ret = (float *) malloc(sizeof(float) * n );

    for( i = 0; i < n ; i++ ) {
      ret[i] = (float) 3.3;
   }

   return ret;
}

void printArray(float *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%f ", arr[i]);

   printf("\n");
}

float cpuReduce(float * h_in, int n)
{
   double total = 0.0;

	 int i;

    for(i = 0; i < n; i ++)
        total += (double) h_in[i];
   // printf("CPU---> %fn", total);
    return total;
}

void reverseIt(float * histogram)
{
	const int n = 60;
  	float d[n];

  	float *d_hist;
  	hipMalloc(&d_hist, n * sizeof(float)); 

	  hipMemcpy(d_hist, histogram, n*sizeof(float), hipMemcpyHostToDevice);


	  staticReverse<<< 1,64 >>>(d_hist, n);
    
	  hipMemcpy(d, d_hist, n*sizeof(float), hipMemcpyDeviceToHost);

    printf("*************************************************\n");

	 for (int i = 0; i < n; i++) {
      printf("%d) %f\n",i, d[i]);
    }

    hipFree(d_hist);
}

//Backprojects M00, M1x, M1y as double pointers, in preparation for reduce summation kernel
void gpuBackProjectMain(unsigned char * hueArray, int hueLength, float * histogram, int width, int xOffset, int yOffset, float ** h_M00, float ** h_M1x, float ** h_M1y)
{
    int tile_width = 64;
    int num_block = ceil(hueLength / (float) tile_width);
    dim3 block(tile_width, 1, 1);
    dim3 grid(num_block, 1, 1);

    const int histogramLength = 60;
    float *d_hist;
    hipError_t err = hipMalloc((void **)&d_hist, histogramLength * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

   err = hipMemcpy(d_hist, histogram, histogramLength * sizeof(float), hipMemcpyHostToDevice);
   if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    unsigned char * d_hueArray;
    err = hipMalloc((void **)&d_hueArray, hueLength * sizeof(unsigned char)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));


    err = hipMemcpy(d_hueArray, hueArray, hueLength * sizeof(unsigned char), hipMemcpyHostToDevice);
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    float * d_M00; //device back projected histogram
    float * d_M1x;
    float * d_M1y;

    err = hipMalloc((void **)&d_M00, hueLength * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    err = hipMalloc((void **)&d_M1x, hueLength * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    err = hipMalloc((void **)&d_M1y, hueLength * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));
    
    gpuBackProjectKernel<<<ceil(hueLength / (float) 64), 64>>>(d_hist, d_hueArray, hueLength, d_M00, d_M1x, d_M1y, width, xOffset, yOffset);

    err = hipMemcpy(*h_M00, d_M00, hueLength * sizeof(float), hipMemcpyDeviceToHost);

    if(err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
    }

   err = hipMemcpy(*h_M1x, d_M1x, hueLength * sizeof(float), hipMemcpyDeviceToHost);

   if(err != hipSuccess)
   {
        printf("%s\n", hipGetErrorString(err));
   }

  err =  hipMemcpy(*h_M1y, d_M1y, hueLength * sizeof(float), hipMemcpyDeviceToHost);


   if(err != hipSuccess)
   {
        printf("%s\n", hipGetErrorString(err));
   }

    hipFree(d_hist);
    hipFree(d_hueArray);
    hipFree(d_M00);
    hipFree(d_M1x);
    hipFree(d_M1y);
}


int gpuReduceMain(int blockWidth, float * M00, float * M1x, float * M1y, int length, int * xc, int * yc)
{
   int tile_width = blockWidth;

   float *h_M00_out, *d_M00_in, *d_M00_out;
   float *h_M1x_out, *d_M1x_in, *d_M1x_out;
   float *h_M1y_out, *d_M1y_in, *d_M1y_out;

   // set up host memory
   h_M00_out = (float *) malloc(length * sizeof(float)); //MAXDRET
   h_M1x_out = (float *) malloc(length * sizeof(float)); //MAXDRET
   h_M1y_out = (float *) malloc(length * sizeof(float)); //MAXDRET

  // memset(h_M00_out, 0, length * sizeof(float)); //MAXDRET
  // memset(h_M1x_out, 0, length * sizeof(float)); //MAXDRET
   //memset(h_M1y_out, 0, length * sizeof(float)); //MAXDRET

   int num_block = ceil(length / (float)tile_width);
   dim3 block(tile_width, 1, 1);
   dim3 grid(num_block, 1, 1);

   // allocate storage for the device
   hipMalloc((void**)&d_M00_in, sizeof(float) * length);
   hipMalloc((void**)&d_M00_out, sizeof(float) * length ); //MAXDRET
   //hipMemset(d_M00_out, 0, sizeof(float) * length ); //MAXDRET

   hipMalloc((void**)&d_M1x_in, sizeof(float) * length);
   hipMalloc((void**)&d_M1x_out, sizeof(float) * length); //MAXDRET
  // hipMemset(d_M1x_out, 0, sizeof(float) * length); //MAXDRET

   hipMalloc((void**)&d_M1y_in, sizeof(float) * length);
   hipMalloc((void**)&d_M1y_out, sizeof(float) * length); //MAXDRET
  // hipMemset(d_M1y_out, 0, sizeof(float) * length); //MAXDRET

   // copy input to the device
   hipMemcpy(d_M00_in, M00, sizeof(float) * length, hipMemcpyHostToDevice);
   hipMemcpy(d_M1x_in, M1x, sizeof(float) * length, hipMemcpyHostToDevice);
   hipMemcpy(d_M1y_in, M1y, sizeof(float) * length, hipMemcpyHostToDevice);

   // time the kernel launches using CUDA events
   hipEvent_t launch_begin, launch_end;
   hipEventCreate(&launch_begin);
   hipEventCreate(&launch_end);

   //----------------------time many kernel launches and take the average time--------------------
   
   int num_in = length, num_out = ceil((float)length / tile_width);
   int launch = 1;

   // record a CUDA event immediately before and after the kernel launch
  hipEventRecord(launch_begin,0);

   while( 1 )
   {
       if(launch % 2 == 1) // odd launch
           gpuSummationReduce<<< grid, block, tile_width * sizeof(float) >>>(d_M00_in, d_M00_out, d_M1x_in, d_M1x_out, d_M1y_in, d_M1y_out, num_in);
       else
           gpuSummationReduce<<< grid, block, tile_width * sizeof(float) >>>(d_M00_out, d_M00_in, d_M1x_out, d_M1x_in, d_M1y_out, d_M1y_in, num_in);

       hipDeviceSynchronize();

       // if the number of local max returned by kernel is greater than the threshold,
       // we do reduction on GPU for these returned local maxes for another pass,
       // until, num_out < threshold

       if(num_out >= THRESH)
       {
           num_in = num_out;
           num_out = ceil((float) num_out / tile_width);
           grid.x = num_out; //change the grid dimension in x direction
       }
       else //copy the ouput of last lauch back to host
       {
           if(launch % 2 == 1)
           {
              hipMemcpy(h_M00_out, d_M00_out, sizeof(float) * num_out, hipMemcpyDeviceToHost);
              hipMemcpy(h_M1x_out, d_M1x_out, sizeof(float) * num_out, hipMemcpyDeviceToHost);
              hipMemcpy(h_M1y_out, d_M1y_out, sizeof(float) * num_out, hipMemcpyDeviceToHost);
           }
           else
           {
              hipMemcpy(h_M00_out, d_M00_in, sizeof(float) * num_out, hipMemcpyDeviceToHost);
              hipMemcpy(h_M1x_out, d_M1x_in, sizeof(float) * num_out, hipMemcpyDeviceToHost);
              hipMemcpy(h_M1y_out, d_M1y_in, sizeof(float) * num_out, hipMemcpyDeviceToHost);
           }

           break;
       }

       launch ++;
   }//end of while

  hipEventRecord(launch_end,0);
  hipEventSynchronize(launch_end);

  // measure the time spent in the kernel
  float time = 0;
  hipEventElapsedTime(&time, launch_begin, launch_end);

 printf("Done! GPU time cost in second: %f\n", time );
 // printf("From GPU: M00 --> %f M1x --> %f M1y --> %f\n", h_M00_out[0], h_M1x_out[0], h_M1y_out[0]);


  //Calculate centroid

  if( h_M00_out[0] > 0){//Can't divide by zero...
        
        *xc = (int) (h_M1x_out[0] /  h_M00_out[0]);
        *yc = (int) (h_M1y_out[0] /  h_M00_out[0]);
        
      //  printf("Inside GPU MeanShift ---> centroid (%d, %d)\n", *xc, *yc);
    }
   printf("**********THIS BETTER BE SO! M00 = %f M1x = %f M1y = %f **************\n", h_M00_out[0], h_M1x_out[0], h_M1y_out[0]);
  //------------------------ now time the sequential code on CPU------------------------------





  clock_t now, then;
  float cpuTotal = 0;

  // timing on CPU
  then = clock();
  cpuTotal = cpuReduce(M00, length);
  now = clock();

  // measure the time spent on CPU
  time = timeCost(then, now);

  printf(" done. CPU time cost in second: %f\n", time * 1000);
  printf("CPU finding total is %f\n", cpuTotal);




  //--------------------------------clean up-----------------------------------------------------
 hipEventDestroy(launch_begin);
 hipEventDestroy(launch_end);

  // deallocate device memory
  hipFree(d_M00_in);
  hipFree(d_M00_out);
  hipFree(d_M1x_in);
  hipFree(d_M1x_out);
  hipFree(d_M1y_in);
  hipFree(d_M1y_out);

  free(h_M00_out);
  free(h_M1x_out);
  free(h_M1y_out);

  return 0;
}


//Backprojects M00, M1x, M1y as double pointers, in preparation for reduce summation kernel
void bpTest(unsigned char * hueArray, int ** convertedArray, int hueLength)
{
	  int index = 40499;
    int s = sizeof(unsigned char);
    printf("Size of unsigned char == %d, hueLength == %d\n", s, hueLength);

    int tile_width = 64;
    int num_block = ceil(hueLength / (float) tile_width);
    dim3 block(tile_width, 1, 1);
    dim3 grid(num_block, 1, 1);

    unsigned char * d_hueArray;

    printf("Checking hueArray--> %d\n", hueArray[index]);

    hipError_t err = hipMalloc((void **) &d_hueArray, hueLength * sizeof(unsigned char)); 

    printf("Checking hueArray--> %d\n", hueArray[index]);
    
    if(err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
    }

    printf("Checking hueArray--> %d\n", hueArray[index]);

    hipMemcpy(d_hueArray, hueArray, hueLength * sizeof(unsigned char), hipMemcpyHostToDevice);

    printf("Checking hueArray--> %d\n", hueArray[index]);

    int * d_converted;

    hipMalloc((void **)&d_converted, hueLength * sizeof(int)); 
   //  hipMemset(d_converted, 0, sizeof(int) * hueLength);

   bpTestKernel<<<ceil(hueLength / (float) 64), 64>>>(d_hueArray, d_converted, hueLength);

   hipMemcpy(*convertedArray, d_converted, hueLength * sizeof(int), hipMemcpyDeviceToHost);
    
   //	hipMemcpy(hueArray, d_hueArray, hueLength * sizeof(unsigned char), hipMemcpyDeviceToHost);


   hipFree(d_converted);
    hipFree(d_hueArray);

}


//***********************************************************************************************//
// Below launches new improved kernel stuff

void mainConstantMemoryHistogramLoad(float * histogram)
{
  setConstantMemoryHistogram(histogram);
}

int launchMeanShiftKernelForSubFrame(unsigned char * hueFrame, int hueFrameLength, int width, int xOffset, int yOffset, int * cx, int * cy)
{
  printf("\nInside Launching GPU MeanShift...\n");

 
   unsigned char * d_in;

    hipError_t err = hipMalloc((void **)&d_in, hueFrameLength * sizeof(unsigned char)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    err = hipMemcpy(d_in, hueFrame, hueFrameLength * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipEvent_t launch_begin, launch_end;
    int tile_width = 1024;
    int num_block = ceil( (float) hueFrameLength / (float) tile_width);
    dim3 block(tile_width, 1, 1);
    dim3 grid(num_block, 1, 1);

    //Make d_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * d_out;
    err = hipMalloc((void **)&d_out, 3 * num_block * sizeof(float)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

      int * readyArray;
    err = hipMalloc((void **)&readyArray, num_block * sizeof(int)); 
    if(err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));
      hipMemset(readyArray, 0, sizeof(int) * num_block ); 

    //Make h_out 3 times the block size to store M00, M1x, M1y values at a stride of num_block
    float * h_out = (float *) malloc(3 * num_block * sizeof(float));

  //  printf("Num_block: %d vs tile_width %d\n", num_block, tile_width);


    if(num_block <= tile_width){

     hipEventCreate(&launch_begin);
     hipEventCreate(&launch_end);

     hipEventRecord(launch_begin,0);

    gpuMeanShiftKernelForSubFrame<<< grid, block >>>(d_in, d_out, readyArray, hueFrameLength, num_block, width, xOffset, yOffset);
      

    err =  hipMemcpy(h_out, d_out, 3 * num_block * sizeof(float), hipMemcpyDeviceToHost);


     hipEventRecord(launch_end,0);
    hipEventSynchronize(launch_end);

    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    printf("GPU time cost in milliseconds for improved meanshift kernel with atomic add: %f\n", time);
    printf("improved meanshift kernel with atomic add total: M00 = %f M1x = %f M1y = %f \n", h_out[0], h_out[num_block], h_out[num_block * 2]);

    //cpuReduce(h_out, num_block);

   //  printArray(h_out, num_block);

  }
  else
    printf("Cannot launch kernel: num_block (%d) > tile_width (%d)\n", num_block, tile_width);


    hipFree(d_out);
    hipFree(readyArray);
    free(h_out);
    hipFree(d_in);

    return 1;
}



































